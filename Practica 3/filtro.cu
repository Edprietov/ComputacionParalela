#include "hip/hip_runtime.h"
//Programa que implementa el algoritmo de aplicación de algunos filtros sobre una imagen usando CUDA
#include <stdio.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Namespaces.
using namespace std;
using namespace cv;

//Filtro que se aplica.
__device__ int filtro(const int *imagen, int x, int y, int alto)
{

	int n = imagen[y + x * alto];
	int rojo = 0, verde = 0, azul = 0, promedio = 0;
	azul += (n % 1000);
	verde += (n / 1000) % 1000;
	rojo += (n / 1000000) % 1000;
	promedio = ((azul + verde + rojo) / 3);
	azul = promedio;
	verde = promedio;
	rojo = promedio;
	return (rojo * 1000000) + (verde * 1000) + azul;
}

//Función que ejecuta cada hilo.
__global__ void hilo_filtro(const int *d_imagen_rgb, const int ancho, const int alto, const int total_hilos, int *d_imagen_filtrada)
{

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int fila_inicial = id * (alto / total_hilos);
	int fila_final = (id + 1) * (alto / total_hilos);
	if (id < alto)
	{
		for (int i = 0; i < ancho; i++)
		{
			for (int j = fila_inicial; j < fila_final; j++)
			{
				d_imagen_filtrada[j + i * alto] = filtro(d_imagen_rgb, i, j, alto);
			}
		}
	}
}

//Prototipos de funciones
Mat lectura_imagen(String nombre_imagen);

int main(int argc, char **argv)
{

	//Variables.
	char *nombre_imagen;
	Mat imagen, imagen_filtrada;
	int num_hilos, num_bloques;

	//Recibir argumentos.
	nombre_imagen = argv[1];
	num_hilos = atoi(argv[2]);

	if (argc != 3)
	{
		cout << "Numero incorrecto de argumentos.\n";
		return -1;
	}

	//Leer imagen
	imagen = lectura_imagen(nombre_imagen);

	//Inicializar variables
	int ancho = imagen.cols;
	int alto = imagen.rows;
	imagen_filtrada = imagen.clone();
	hipError_t err = hipSuccess;

	//Malloc host
	int num_elementos = ancho * alto;
	size_t size = num_elementos * sizeof(int);
	int *h_imagen_rgb = (int *)malloc(size);
	int *h_imagen_filtrada = (int *)malloc(size);

	//Imagen a un vector 3D
	int aux = 0;
	for (int i = 0; i < ancho; i++)
	{
		for (int j = 0; j < alto; j++)
		{
			h_imagen_rgb[aux] = imagen.at<Vec3b>(j, i)[0];
			h_imagen_rgb[aux] += imagen.at<Vec3b>(j, i)[1] * 1000;
			h_imagen_rgb[aux] += imagen.at<Vec3b>(j, i)[2] * 1000000;
			aux++;
		}
	}

	//Reserva de memoria en el dispositivo
	//Imagen
	int *d_imagen_rgb = NULL;
	err = hipMalloc((void **)&d_imagen_rgb, size);
	if (err != hipSuccess)
	{
		cout << "Error separando espacio imagen normal en GPU " << hipGetErrorString(err) << endl;
		return -1;
	}
	//Resultado
	int *d_imagen_filtrada = NULL;
	err = hipMalloc((void **)&d_imagen_filtrada, size);
	if (err != hipSuccess)
	{
		cout << "Error separando espacio imagen filtrada en GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Copia de imagen desde el host al dispositivo
	err = hipMemcpy(d_imagen_rgb, h_imagen_rgb, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Lanzar kernel GPU
    num_bloques = (alto + num_hilos - 1) / num_hilos;
	hilo_filtro<<<num_bloques, num_hilos>>>(d_imagen_rgb, ancho, alto, alto, d_imagen_filtrada);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Copiar de GPU a CPU
	cout << "Copiando datos desde la GPU a CPU." << endl;
	err = hipMemcpy(h_imagen_filtrada, d_imagen_filtrada, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		cout << "Error copiando desde GPU a CPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Escribir imagen difuminada.
	aux = 0;
	for (int i = 0; i < ancho; i++)
	{
		for (int j = 0; j < alto; j++)
		{
			imagen_filtrada.at<Vec3b>(j, i)[0] = (unsigned char)((h_imagen_filtrada[aux]) % 1000);
			imagen_filtrada.at<Vec3b>(j, i)[1] = (unsigned char)((h_imagen_filtrada[aux] / 1000) % 1000);
			imagen_filtrada.at<Vec3b>(j, i)[2] = (unsigned char)((h_imagen_filtrada[aux] / 1000000) % 1000);
			aux++;
		}
	}
	//String nombre_archivo = "./Resultados/filtro" + to_string(numfiltro) + "__nombre_ " + nombre;
	String nombre_archivo = "./Resultados/filtrada_";
	imwrite(nombre_archivo += nombre_imagen, imagen_filtrada);

	//Liberar espacio
	err = hipFree(d_imagen_rgb);
	if (err != hipSuccess)
	{
		cout << "Error liberando memoria de imagen normal " << hipGetErrorString(err) << endl;
		return -1;
	}

	err = hipFree(d_imagen_filtrada);
	if (err != hipSuccess)
	{
		cout << "Error liberando memoria de imagen difuminada " << hipGetErrorString(err) << endl;
		return -1;
	}

	free(h_imagen_rgb);
	free(h_imagen_filtrada);
	return 0;
}

/*****Procedimiento que lee la imagen******/
Mat lectura_imagen(String nombre_imagen)
{
    // Lectura de la imagen
    Mat imagen = imread("./Assets/"+nombre_imagen, 1);

    // Manejo de error en caso de que no sea encontrada la imagen
    if (imagen.empty())
    {
        cout << "Archivo de imagen "
             << "No encontrado" << endl;
        cin.get();
        return imagen;
    }
    return imagen;
}

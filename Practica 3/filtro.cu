#include "hip/hip_runtime.h"
//Programa que implementa el algoritmo de aplicación de algunos filtros sobre una imagen usando CUDA
#include <stdio.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Namespaces.
using namespace std;
using namespace cv;

//Filtro que se aplica.
__device__ bool aplicar_filtro(int rojo, int verde, int azul, int filtro_a_aplicar)
{

	switch (filtro_a_aplicar)
	{
		//FILTRO AMARILLO
	case 1:
		if (rojo > 200 && verde > 100 && azul < 85)
		{
			return true;
		}
		break;
		//FILTRO AZUL
	case 2:
		if (rojo < 80 && verde > 130 && azul > 170)
		{
			return true;
		}
		break;
		//FILTRO VERDE
	case 3:
		if (rojo < 91 && verde > 159 && azul < 91)
		{
			return true;
		}
		break;
	default:
		break;
	}
	return false;
}

__device__ int filtro(const int *imagen, int x, int y, int alto, int num_filtro)
{

	int n = imagen[y + x * alto];
	int rojo = 0, verde = 0, azul = 0, promedio = 0;
	azul += (n % 1000);
	verde += (n / 1000) % 1000;
	rojo += (n / 1000000) % 1000;
	promedio = ((azul + verde + rojo) / 3);
	if (!aplicar_filtro(rojo, verde, azul, num_filtro))
	{
		azul = promedio;
		verde = promedio;
		rojo = promedio;
	}
	return (rojo * 1000000) + (verde * 1000) + azul;
}

//Función que ejecuta cada hilo.
__global__ void hilo_filtro(const int *d_imagen_rgb, const int ancho, const int alto, const int total_hilos, int *d_imagen_filtrada, int num_filtro)
{

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int fila_inicial = id * (alto / total_hilos);
	int fila_final = (id + 1) * (alto / total_hilos);
	if (id < alto)
	{
		for (int i = 0; i < ancho; i++)
		{
			for (int j = fila_inicial; j < fila_final; j++)
			{
				d_imagen_filtrada[j + i * alto] = filtro(d_imagen_rgb, i, j, alto, num_filtro);
			}
		}
	}
}

//Prototipos de funciones
Mat lectura_imagen(String nombre_imagen);

int main(int argc, char **argv)
{

	//Variables.
	char *nombre_imagen;
	Mat imagen, imagen_filtrada;
	int num_hilos, num_bloques, num_filtro, iteracion;

	//Recibir argumentos.
	nombre_imagen = argv[1];
	num_hilos = atoi(argv[2]);
	num_filtro = atoi(argv[3]);
	iteracion = atoi(argv[4]);

	if (argc != 5)
	{
		cout << "Numero incorrecto de argumentos.\n";
		return -1;
	}

	//Leer imagen
	imagen = lectura_imagen(nombre_imagen);

	//Inicializar variables
	int ancho = imagen.cols;
	int alto = imagen.rows;
	imagen_filtrada = imagen.clone();
	hipError_t err = hipSuccess;

	//Malloc host
	int num_elementos = ancho * alto;
	size_t size = num_elementos * sizeof(int);
	int *h_imagen_rgb = (int *)malloc(size);
	int *h_imagen_filtrada = (int *)malloc(size);

	//Imagen a un vector 3D
	int aux = 0;
	for (int i = 0; i < ancho; i++)
	{
		for (int j = 0; j < alto; j++)
		{
			h_imagen_rgb[aux] = imagen.at<Vec3b>(j, i)[0];
			h_imagen_rgb[aux] += imagen.at<Vec3b>(j, i)[1] * 1000;
			h_imagen_rgb[aux] += imagen.at<Vec3b>(j, i)[2] * 1000000;
			aux++;
		}
	}

	//Reserva de memoria en el dispositivo
	//Imagen
	int *d_imagen_rgb = NULL;
	err = hipMalloc((void **)&d_imagen_rgb, size);
	if (err != hipSuccess)
	{
		cout << "Error separando espacio imagen normal en GPU " << hipGetErrorString(err) << endl;
		return -1;
	}
	//Resultado
	int *d_imagen_filtrada = NULL;
	err = hipMalloc((void **)&d_imagen_filtrada, size);
	if (err != hipSuccess)
	{
		cout << "Error separando espacio imagen filtrada en GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Copia de imagen desde el host al dispositivo
	err = hipMemcpy(d_imagen_rgb, h_imagen_rgb, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Lanzar kernel GPU
	num_bloques = (alto + num_hilos - 1) / num_hilos;
	hilo_filtro<<<num_bloques, num_hilos>>>(d_imagen_rgb, ancho, alto, alto, d_imagen_filtrada, num_filtro);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Copiar de GPU a CPU
	err = hipMemcpy(h_imagen_filtrada, d_imagen_filtrada, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		cout << "Error copiando desde GPU a CPU " << hipGetErrorString(err) << endl;
		return -1;
	}

	//Escribir imagen difuminada.
	aux = 0;
	for (int i = 0; i < ancho; i++)
	{
		for (int j = 0; j < alto; j++)
		{
			imagen_filtrada.at<Vec3b>(j, i)[0] = (unsigned char)((h_imagen_filtrada[aux]) % 1000);
			imagen_filtrada.at<Vec3b>(j, i)[1] = (unsigned char)((h_imagen_filtrada[aux] / 1000) % 1000);
			imagen_filtrada.at<Vec3b>(j, i)[2] = (unsigned char)((h_imagen_filtrada[aux] / 1000000) % 1000);
			aux++;
		}
	}

	if (num_hilos == 1024 && iteracion == 15)
	{
		String nombre_archivo = "./Resultados/filtro_" + to_string(num_filtro) + "_";
		imwrite(nombre_archivo += nombre_imagen, imagen_filtrada);
	}

	//Liberar espacio
	err = hipFree(d_imagen_rgb);
	if (err != hipSuccess)
	{
		cout << "Error liberando memoria de imagen normal " << hipGetErrorString(err) << endl;
		return -1;
	}

	err = hipFree(d_imagen_filtrada);
	if (err != hipSuccess)
	{
		cout << "Error liberando memoria de imagen difuminada " << hipGetErrorString(err) << endl;
		return -1;
	}

	free(h_imagen_rgb);
	free(h_imagen_filtrada);
	return 0;
}

/*****Procedimiento que lee la imagen******/
Mat lectura_imagen(String nombre_imagen)
{
	// Lectura de la imagen
	Mat imagen = imread("./Assets/" + nombre_imagen, 1);

	// Manejo de error en caso de que no sea encontrada la imagen
	if (imagen.empty())
	{
		cout << "Archivo de imagen "
			 << "No encontrado" << endl;
		cin.get();
		return imagen;
	}
	return imagen;
}

#include "hip/hip_runtime.h"
//Programa que implementa el algoritmo de aplicación de algunos filtros sobre una imagen
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <fstream>

using namespace cv;
using namespace std;


__global__
void saxpy()
{

}

//Prototipos de funciones
Mat lectura_imagen(String nombre_imagen);
void filtro_amarillo(String nombre);
ofstream archivo;


int main(int argc, char **argv)
{
    filtro_amarillo(argv[1]);
    waitKey(0);
    return 0;
}

//***filtro2*****
void filtro_amarillo(String nombre)
{
    int pixel = 0;
    unsigned tiempo_final, tiempo_inicial;
    Mat imagen ;

        imagen = lectura_imagen(nombre);
        tiempo_inicial = clock();

        for (int y = 0; y < imagen.rows; y++)
        {
            for (int x = 0; x < imagen.cols; x++)
            {
                Vec3b color = imagen.at<Vec3b>(y, x);
                int azul = (int)color.val[0];
                int verde = (int)color.val[1];
                int rojo = (int)color.val[2];
                int promedio = (int)((azul + verde + rojo) / 3);

                color.val[0] = promedio;
                color.val[1] = promedio;
                color.val[2] = promedio;
             
                imagen.at<Vec3b>(y, x) = color;
            }
        }
    imwrite("filtradacolor" + nombre, imagen);
}

//*****Procedimiento que lee la imagen******
Mat lectura_imagen(String nombre_imagen)
{
    // Lectura de la imagen
    Mat imagen = imread("./../Assets/"+nombre_imagen, 1);
    // Manejo de error en caso de que no sea encontrada la imagen
    if (imagen.empty())
    {
        cout << "Archivo de imagen "
             << "No encontrado" << endl;
        cin.get();
        return imagen;
    }
    return imagen;
}
